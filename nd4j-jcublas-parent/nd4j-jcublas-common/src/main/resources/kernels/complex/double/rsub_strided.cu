extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void rsub_strided_double(int n,int xOffset,int yOffset, double *dx, double *dy,int incx,int incy,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                          if(i >= xOffset && i >= yOffset &&  i % incx == 0 && i % incy == 0)
                                result[i] = dx[i] / dy[i];
             }

 }

